#include "hip/hip_runtime.h"
#include <vector>
#include <hipcub/hipcub.hpp>
#include <cuda/std/semaphore>
#include <torch/extension.h>

using semaphore = cuda::std::counting_semaphore<>;
const int BLOCK_THREADS = 512;


template <int BLOCK_THREADS, typename scalar_t> 
__device__  scalar_t dot(scalar_t *a, scalar_t *b, int length, int tx){
    typedef hipcub::BlockReduce<scalar_t, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int loop_times = ceil((float)length / (float)BLOCK_THREADS);
    __shared__ scalar_t dot;
    if(tx == 0) dot = 0;
    __syncthreads();

    for(int i = 0; i < loop_times; ++i){
        int idx = i * BLOCK_THREADS + tx;
        scalar_t prod = 0;
        if(idx < length) prod = a[idx] * b[idx];
        scalar_t reduce = BlockReduce(temp_storage).Sum(prod);

        if(tx == 0) dot += reduce;
        __syncthreads();
    }

    return dot;
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__ void reflections(scalar_t *R, scalar_t *vs, int m, int n, semaphore* *sems){ //vs still float precision?
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int v_len = n - bx;
    scalar_t *v = &vs[bx * n + bx];

    if(tx == 0)
        sems[bx * m + bx]->acquire();
    __syncthreads();

    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] = - R[bx * n + bx + idx];

    scalar_t norm_v_sq = dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx);
    if(tx == 0) v[0] += copysign(sqrt(norm_v_sq), v[0]);
    
    scalar_t norm_v = sqrt(dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx));
    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] /= norm_v;

    for(int row = 0; row < m; ++row){ //dynamic parallelsim and avoid this loop?
        if(row > bx){
            if(tx == 0) sems[bx * m + row]->acquire();   
            __syncthreads();
        }     

        scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(&R[row * n + bx], v, v_len, tx);
        
        for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
            R[row * n + bx + idx] -= 2.0 * v[idx] * dot_value;

        if(row > bx){
            __syncthreads();
            if (tx == 0) sems[(bx + 1) * m + row]->release();
        }
    }
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__  void Q_loop(scalar_t *Q, scalar_t *vs, int n, int col){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    scalar_t *v = &vs[col * n];
    
    scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(v, &Q[bx * n], n, tx);

    for(int idx = tx; idx < n; idx += BLOCK_THREADS)
        Q[bx * n + idx] -= 2.0 * v[idx] * dot_value;
}

template <typename scalar_t> 
__global__ void add_diag(scalar_t *A, int n, scalar_t value){
    int tx = threadIdx.x;
    A[tx * n + tx] += value;
}

__global__ 
void init_sems(semaphore* *sems, int m){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    hipMalloc(&sems[bx * m + tx], sizeof(semaphore*));
    semaphore sem(1);
    sems[bx * m + tx] = &sem;
    sem.release();
}


template <typename scalar_t> 
void dispatched_implementation(torch::Tensor A, torch::Tensor Q, int m, int n, float epsilon){
    scalar_t eps = (scalar_t) epsilon;
    add_diag<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, eps);
    
    scalar_t *vs;
    hipMalloc(&vs, m * n * sizeof(scalar_t));
    hipMemset(vs, 0, m * n * sizeof(scalar_t));

    semaphore* *sems = new semaphore*[(m+1)*m];
    init_sems<<<m + 1, m>>>(sems, m);
    hipDeviceSynchronize();

    reflections<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS>>>(A.data<scalar_t>(), vs, m, n, sems);
    
    hipMemset(Q.data<scalar_t>(), 0, m * n * sizeof(scalar_t));
    add_diag<scalar_t><<<1, m>>>(Q.data<scalar_t>(), n, 1);
    hipDeviceSynchronize();

    //dim3 blockDim = dim3(m, m);
    for(int col = m - 1; col >= 0; --col){
        Q_loop<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS>>>(Q.data<scalar_t>(), vs, n, col);
        hipDeviceSynchronize();
    }

    hipFree(sems);
    hipFree(vs);
}

void qr_orthogonalization_cuda(torch::Tensor A, torch::Tensor Q, int m, int n, float epsilon){
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    A.scalar_type(), "qr_orthogonalization_cuda", ([&] {
        dispatched_implementation<scalar_t>(A, Q, m, n, epsilon);
    }));
}