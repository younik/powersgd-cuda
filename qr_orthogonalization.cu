#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <cuda/std/semaphore>
#include <torch/extension.h>

using semaphore = cuda::std::counting_semaphore<>;
const int BLOCK_THREADS = 512;


template <int BLOCK_THREADS, typename scalar_t>
__device__  scalar_t dot(scalar_t *a, scalar_t *b, int length, int tx){
    typedef hipcub::BlockReduce<scalar_t, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    int loopTimes = ceil((float)length / (float)BLOCK_THREADS);
    __shared__ scalar_t dot;
    if(tx == 0) dot = 0;
    __syncthreads();

    for(int i = 0; i < loopTimes; ++i){
        int idx = i * BLOCK_THREADS + tx;

        scalar_t prod = (idx < length)? a[idx] * b[idx] : (scalar_t) 0;
        scalar_t reduce = BlockReduce(tmpStorage).Sum(prod);

        if(tx == 0) dot += reduce;
        __syncthreads();
    }

    return dot;
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__ void reflections(scalar_t *R, scalar_t *vs, int m, int n, semaphore *sems){ //vs still float precision?
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int vLen = n - bx;
    scalar_t *v = &vs[bx * n + bx];

    if(tx == 0)
        sems[bx * m + bx].acquire();
    __syncthreads();

    for(int idx = tx; idx < vLen; idx += BLOCK_THREADS)
        v[idx] = - R[bx * n + bx + idx];

    scalar_t normVSq = dot<BLOCK_THREADS, scalar_t>(v, v, vLen, tx);
    if(tx == 0) v[0] += copysign(sqrt(normVSq), v[0]);
    
    scalar_t normV = sqrt(dot<BLOCK_THREADS, scalar_t>(v, v, vLen, tx));
    for(int idx = tx; idx < vLen; idx += BLOCK_THREADS)
        v[idx] /= normV;

    for(int row = 0; row < m; ++row){ //dynamic parallelsim and avoid this loop?
        if(row > bx){
            if(tx == 0) sems[bx * m + row].acquire();   
            __syncthreads();
        }     

        scalar_t dotValue = dot<BLOCK_THREADS, scalar_t>(&R[row * n + bx], v, vLen, tx);
        
        for(int idx = tx; idx < vLen; idx += BLOCK_THREADS)
            R[row * n + bx + idx] -= 2.0 * v[idx] * dotValue;

        if(row > bx){
            __syncthreads();
            if (tx == 0) sems[(bx + 1) * m + row].release();
        }
    }
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__  void QLoop(scalar_t *Q, scalar_t *vs, int n, int m, semaphore *sems){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int vIdx = m - blockIdx.y - 1;
    scalar_t *v = &vs[vIdx * n];

    if(tx==0) sems[(vIdx + 1) * m + bx].acquire();
    __syncthreads();
    
    scalar_t dotValue = dot<BLOCK_THREADS, scalar_t>(v, &Q[bx * n], n, tx);

    for(int idx = tx; idx < n; idx += BLOCK_THREADS)
        Q[bx * n + idx] -= 2.0 * v[idx] * dotValue;

    __syncthreads();
    if(tx==0) sems[vIdx  * m + bx].release();
}

template <typename scalar_t> 
__global__ void addDiagonal(scalar_t *A, int n, scalar_t value){
    int tx = threadIdx.x;
    A[tx * n + tx] += value;
}

__global__ 
void initSems(semaphore *sems, int m){
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    new (&sems[bx * m + tx]) semaphore (bx == 0);
}

__global__
void releaseSems(semaphore *sems){
    sems[threadIdx.x].release();
}

template <typename scalar_t> 
void dispatchedImplementation(torch::Tensor A, int m, int n, float epsilon){
    semaphore *sems;
    hipMalloc((void**)&sems, (m + 1) * m * sizeof(semaphore));
    initSems<<<m + 1, m>>>(sems, m);
    
    torch::Tensor vs = torch::zeros_like(A);

    addDiagonal<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, (scalar_t) epsilon);

    hipDeviceSynchronize();
    reflections<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS>>>(A.data<scalar_t>(), vs.data<scalar_t>(), m, n, sems);
    hipDeviceSynchronize();

    releaseSems<<<1, m>>>(&sems[m*m]);
    hipMemset(A.data<scalar_t>(), 0, m * n * sizeof(scalar_t));
    addDiagonal<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, 1);
    
    hipDeviceSynchronize();

    dim3 blockDim = dim3(m, m);
    QLoop<BLOCK_THREADS, scalar_t><<<blockDim, BLOCK_THREADS>>>(A.data<scalar_t>(), vs.data<scalar_t>(), n, m, sems);
    hipDeviceSynchronize();

    hipFree(sems);
}

void qrOrthogonalizationCuda(torch::Tensor A, int m, int n, float epsilon){
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    A.scalar_type(), "qr_orthogonalization_cuda", ([&] {
        dispatchedImplementation<scalar_t>(A, m, n, epsilon);
    }));
}