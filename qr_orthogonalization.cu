#include "hip/hip_runtime.h"
#include <vector>
#include <hipcub/hipcub.hpp>
#include <cuda/std/semaphore>
#include <torch/extension.h>

using semaphore = cuda::std::counting_semaphore<>;


template <int BLOCK_THREADS, typename scalar_t> 
__device__  scalar_t dot(scalar_t *a, scalar_t *b, int length, int tx){
    typedef hipcub::BlockReduce<scalar_t, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int loop_times = ceil((float)length / (float)BLOCK_THREADS);
    __shared__ scalar_t dot;
    if(tx == 0) dot = 0;
    __syncthreads();

    for(int i = 0; i < loop_times; ++i){
        int idx = i * BLOCK_THREADS + tx;
        scalar_t prod = 0;
        if(idx < length) prod = a[idx] * b[idx];
        scalar_t reduce = BlockReduce(temp_storage).Sum(prod); //not all thread calls when length not power of 2, ok?

        if(tx == 0) dot += reduce;
        __syncthreads();
    }

    return dot;
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__ void reflections(scalar_t *R, scalar_t *vs, int m, int n, semaphore *sems){ //vs still float precision?
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int v_len = n - bx;
    // extern __shared__ __align__(sizeof(scalar_t)) unsigned char ref_vec[];
    // scalar_t *v = reinterpret_cast<scalar_t *>(ref_vec);
    scalar_t *v = &vs[bx * n + bx];

    if(tx == 0)
        sems[bx * m + bx].acquire();
    __syncthreads();

    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] = - R[bx * n + bx + idx];

    scalar_t norm_v_sq = dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx);
    if(tx == 0) v[0] += copysign(sqrt(norm_v_sq), v[0]);
    
    __syncthreads();

    scalar_t norm_v = sqrt(dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx));
    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] /= norm_v;

    for(int row = 0; row < m; ++row){ //dynamic parallelsim and avoid this loop?
        if(row > bx){
            if(tx == 0) sems[bx * m + row].acquire();   
            __syncthreads();
        }     

        scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(&R[row * n + bx], v, v_len, tx);
        
        for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
            R[row * n + bx + idx] -= 2.0 * v[idx] * dot_value;

        if(row > bx){
            __syncthreads();
            if (tx == 0) sems[(bx + 1) * m + row].release();
        }
    }

    // for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
    //     vs[bx * m + bx + idx] = v[idx]; //TODO: vs not matrix, variable row size
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__  void Q_loop(scalar_t *Q, scalar_t *vs, int n, int col){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    scalar_t *v = &vs[col * n];
    
    scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(v, &Q[bx * n], n, tx);

    for(int idx = tx; idx < n; idx += BLOCK_THREADS)
        Q[bx * n + idx] -= 2.0 * v[idx] * dot_value;
}

template <typename scalar_t> 
__global__ void add_diag(scalar_t *A, int n, scalar_t value){
    int tx = threadIdx.x;
    A[tx * n + tx] += value;
}


template <typename scalar_t> 
void dispatched_implementation(torch::Tensor A, torch::Tensor Q, int m, int n, float epsilon){
    scalar_t eps = (scalar_t) epsilon;
    add_diag<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, eps);
    
    scalar_t *vs;
    hipMalloc(&vs, m * n * sizeof(scalar_t));
    hipMemset(vs, 0, m * n * sizeof(scalar_t));

    semaphore *sems;
    hipMalloc(&sems, m * (m + 1) * sizeof(semaphore));
    for(int i=0; i<m; ++i){ //init on device?
        semaphore sem_h(1);
        hipMemcpyAsync(&sems[i], &sem_h, sizeof(semaphore), hipMemcpyHostToDevice);
    }
    for(int i=m; i<m*(m+1); ++i){
        semaphore sem_h(0);
        hipMemcpyAsync(&sems[i], &sem_h, sizeof(semaphore), hipMemcpyHostToDevice);
    }

    hipDeviceSynchronize();

    reflections<1024, scalar_t><<<m, n>>>(A.data<scalar_t>(), vs, m, n, sems);
    
    hipMemset(Q.data<scalar_t>(), 0, m * n * sizeof(scalar_t));
    add_diag<scalar_t><<<1, m>>>(Q.data<scalar_t>(), n, 1);
    hipDeviceSynchronize();

    for(int col = m - 1; col >= 0; --col){
        Q_loop<1024, scalar_t><<<m, n>>>(Q.data<scalar_t>(), vs, n, col);
        hipDeviceSynchronize();
    }

    hipFree(sems);
    hipFree(vs);
}

void qr_orthogonalization_cuda(torch::Tensor A, torch::Tensor Q, int m, int n, float epsilon){
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    A.scalar_type(), "qr_orthogonalization_cuda", ([&] {
        dispatched_implementation<scalar_t>(A, Q, m, n, epsilon);
    }));
}
