#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <cuda/std/semaphore>
#include <torch/extension.h>

using semaphore = cuda::std::counting_semaphore<>;
const int BLOCK_THREADS = 512;


template <int BLOCK_THREADS, typename scalar_t>
__device__  scalar_t dot(scalar_t *a, scalar_t *b, int length, int tx){
    typedef hipcub::BlockReduce<scalar_t, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int loop_times = ceil((float)length / (float)BLOCK_THREADS);
    __shared__ scalar_t dot;
    if(tx == 0) dot = 0;
    __syncthreads();

    for(int i = 0; i < loop_times; ++i){
        int idx = i * BLOCK_THREADS + tx;
        scalar_t prod = 0;
        if(idx < length) prod = a[idx] * b[idx];
        
        scalar_t reduce = BlockReduce(temp_storage).Sum(prod);

        if(tx == 0) dot += reduce;
        __syncthreads();
    }

    return dot;
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__ void reflections(scalar_t *R, scalar_t *vs, int m, int n, semaphore *sems){ //vs still float precision?
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int v_len = n - bx;
    scalar_t *v = &vs[bx * n + bx];

    if(tx == 0)
        sems[bx * m + bx].acquire();
    __syncthreads();

    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] = - R[bx * n + bx + idx];

    scalar_t norm_v_sq = dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx);
    if(tx == 0) v[0] += copysign(sqrt(norm_v_sq), v[0]);
    
    scalar_t norm_v = sqrt(dot<BLOCK_THREADS, scalar_t>(v, v, v_len, tx));
    for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
        v[idx] /= norm_v;

    for(int row = 0; row < m; ++row){ //dynamic parallelsim and avoid this loop?
        if(row > bx){
            if(tx == 0) sems[bx * m + row].acquire();   
            __syncthreads();
        }     

        scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(&R[row * n + bx], v, v_len, tx);
        
        for(int idx = tx; idx < v_len; idx += BLOCK_THREADS)
            R[row * n + bx + idx] -= 2.0 * v[idx] * dot_value;

        if(row > bx){
            __syncthreads();
            if (tx == 0) sems[(bx + 1) * m + row].release();
        }
    }
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__  void Q_loop(scalar_t *Q, scalar_t *vs, int n, int m, semaphore *sems){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int v_idx = m - blockIdx.y - 1;
    scalar_t *v = &vs[v_idx * n];

    if(tx==0) sems[(v_idx + 1) * m + bx].acquire();
    __syncthreads();
    
    scalar_t dot_value = dot<BLOCK_THREADS, scalar_t>(v, &Q[bx * n], n, tx);

    for(int idx = tx; idx < n; idx += BLOCK_THREADS)
        Q[bx * n + idx] -= 2.0 * v[idx] * dot_value;

    __syncthreads();
    if(tx==0) sems[v_idx  * m + bx].release();
}

template <typename scalar_t> 
__global__ void add_diag(scalar_t *A, int n, scalar_t value){
    int tx = threadIdx.x;
    A[tx * n + tx] += value;
}

__global__ 
void init_sems(semaphore *sems, int m){
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    new (&sems[bx * m + tx]) semaphore (bx == 0);
}

__global__
void release_sems(semaphore *sems){
    sems[threadIdx.x].release();
}

template <typename scalar_t> 
void dispatched_implementation(torch::Tensor A, int m, int n, float epsilon){
    semaphore *sems;
    hipMalloc((void**)&sems, (m + 1) * m * sizeof(semaphore));
    init_sems<<<m + 1, m>>>(sems, m);
    
    scalar_t *vs;
    hipMalloc(&vs, m * n * sizeof(scalar_t));
    hipMemsetAsync(vs, 0, m * n * sizeof(scalar_t));

    scalar_t eps = (scalar_t) epsilon;
    add_diag<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, eps);

    hipDeviceSynchronize();
    reflections<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS>>>(A.data<scalar_t>(), vs, m, n, sems);
    hipDeviceSynchronize();

    release_sems<<<1, m>>>(&sems[m*m]);
    hipMemset(A.data<scalar_t>(), 0, m * n * sizeof(scalar_t));
    add_diag<scalar_t><<<1, m>>>(A.data<scalar_t>(), n, 1);
    
    hipDeviceSynchronize();

    dim3 blockDim = dim3(m, m);
    Q_loop<BLOCK_THREADS, scalar_t><<<blockDim, BLOCK_THREADS>>>(A.data<scalar_t>(), vs, n, m, sems);
    hipDeviceSynchronize();

    hipFree(sems);
    hipFree(vs);
}

void qr_orthogonalization_cuda(torch::Tensor A, int m, int n, float epsilon){
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    A.scalar_type(), "qr_orthogonalization_cuda", ([&] {
        dispatched_implementation<scalar_t>(A, m, n, epsilon);
    }));
}