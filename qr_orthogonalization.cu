#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <cuda/std/semaphore>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

using semaphore = cuda::std::counting_semaphore<>;

__device__ __forceinline__ void wait_barrier(int* barrier, int target){
    if (threadIdx.x == 0){
        int counter;
        do {
            asm volatile ("ld.relaxed.gpu.global.s32 %0, [%1];" : "=r"(counter): "l"(barrier) );
        }
        while (counter < target);
    }
    __syncthreads();
}

template <int BLOCK_THREADS, typename scalar_t>
__device__  __forceinline__ scalar_t dot(scalar_t *a, scalar_t *b, uint length, int tx){
    typedef hipcub::BlockReduce<scalar_t, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    uint unroll = ceil( (float)length / (float)BLOCK_THREADS );
    uint idx = (tx & -32u)*unroll + (tx & 31);

    scalar_t localProd = 0;
    for (int i = 0; i < unroll; ++i){
        localProd += (idx < length)? a[idx] * b[idx] : (scalar_t) 0;
        idx += 32;
    }

    __shared__ scalar_t dot;
    scalar_t reduce = BlockReduce(tmpStorage).Sum(localProd);

    if (tx == 0) 
        dot = reduce;
    __syncthreads();

    return dot;
}

template <int BLOCK_THREADS, typename scalar_t> 
__global__ void reflections(scalar_t *R, scalar_t *vs, int m, int n, int *barrier){
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    for (int row = 0; row < bx; ++row){
        wait_barrier(barrier, row);

        scalar_t *v = &vs[row * n + row];
        uint vLen = n - row;
        scalar_t dotValue = dot<BLOCK_THREADS, scalar_t>(&R[bx * n + row], v, vLen, tx);
        
        for (uint idx = tx; idx < vLen; idx += BLOCK_THREADS)
            R[bx * n + row + idx] -= 2.0 * v[idx] * dotValue;
    }

    scalar_t *v = &vs[bx * n + bx];
    int vLen = n - bx;
    for (uint idx = tx; idx < vLen; idx += BLOCK_THREADS)
        v[idx] = - R[bx * n + bx + idx];

    scalar_t normVSq = dot<BLOCK_THREADS, scalar_t>(v, v, vLen, tx);
    if (tx == 0) 
        v[0] += copysign(sqrt(normVSq), v[0]);
    
    scalar_t normV = sqrt(dot<BLOCK_THREADS, scalar_t>(v, v, vLen, tx));
    for (uint idx = tx; idx < vLen; idx += BLOCK_THREADS)
        v[idx] /= normV;

    __syncthreads();
    if(tx == 0)
        asm volatile ("st.global.cg.s32 [%0], %1;" :: "l"(barrier), "r"(bx));

}

template <int BLOCK_THREADS, typename scalar_t> 
__global__  void QLoop(scalar_t *Q, scalar_t *vs, int n, int m){
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    for (int vIdx = 0; vIdx < m; ++vIdx){
        scalar_t *v = &vs[vIdx * n + vIdx];
        uint vLen = n - vIdx;
    
        scalar_t dotValue = dot<BLOCK_THREADS, scalar_t>(v, &Q[bx * n + vIdx], vLen, tx);

        for (uint idx = tx; idx < vLen ; idx += BLOCK_THREADS)
            Q[bx * n + vIdx + idx] -= 2.0 * v[idx] * dotValue;
    }
}

template <int BLOCK_THREADS, typename scalar_t> 
void qrMain(torch::Tensor A, int m, int n, float epsilon){
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    auto options = torch::TensorOptions().dtype(torch::kInt32).device(A.device());
    torch::Tensor barrier = torch::zeros({1}, options);
    
    torch::Tensor vs = torch::zeros_like(A);
    A.diagonal().add_((scalar_t) epsilon);
    
    reflections<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS, 0, stream>>>(A.data<scalar_t>(), vs.data<scalar_t>(), m, n, barrier.data<int>());

    A.fill_(0);
    A.fill_diagonal_(1);
    QLoop<BLOCK_THREADS, scalar_t><<<m, BLOCK_THREADS, 0, stream>>>(A.data<scalar_t>(), vs.data<scalar_t>(), n, m);
}

template <typename scalar_t> 
void typedImplementation(torch::Tensor A, int m, int n, float epsilon){
    if (n < 512)
        return qrMain<256, scalar_t>(A, m, n, epsilon);
    else if (n < 1024)
        return qrMain<512, scalar_t>(A, m, n, epsilon);
    else
        return qrMain<1024, scalar_t>(A, m, n, epsilon);
}

void qrOrthogonalizationCuda(torch::Tensor A, int m, int n, float epsilon){
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
    A.scalar_type(), "qr_orthogonalization_cuda", ([&] {
        typedImplementation<scalar_t>(A, m, n, epsilon);
    }));
}